#include <iostream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>
#include <chrono>
#include <random>

// CUDA kernel for ReLU derivative
__global__ void relu_derivative_kernel(const float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = (input[idx] > 0) ? 1.0f : 0.0f;
    }
}

// CUDA kernel for computing gradients of the output layer
__global__ void output_layer_gradients_kernel(const float* output, const int* labels, float* grad_output, int size, int num_classes) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        int sample_idx = idx / num_classes;
        int label = labels[sample_idx];
        grad_output[idx] = output[idx] - (idx % num_classes == label ? 1.0f : 0.0f);
    }
}


