#include <iostream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>
#include <chrono>
#include <random>

// CUDA kernel for KL divergence
__global__ void kl_divergence_kernel(const float* P, const float* Q, float* divergence, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float p = P[idx];
        float q = Q[idx];
        divergence[idx] = p * logf(p / q);
    }
}

// CPU reference implementation of KL divergence
float kl_divergence_cpu(const std::vector<float>& P, const std::vector<float>& Q, int size) {
    float divergence = 0.0f;
    for (int i = 0; i < size; i++) {
        divergence += P[i] * logf(P[i] / Q[i]);
    }
    return divergence;
}

// Validation function
bool validate_results(float cpu_divergence, float gpu_divergence, float tolerance = 1e-5) {
    if (std::fabs(cpu_divergence - gpu_divergence) > tolerance) {
        std::cout << "Mismatch: CPU=" << cpu_divergence << ", GPU=" << gpu_divergence << std::endl;
        return false;
    }
    return true;
}

