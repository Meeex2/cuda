#include "hip/hip_runtime.h"
%%writefile flash.cu
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <chrono>
#include <vector>
#include <cstdlib>
#include <cstdio>

#define CUDA_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort=true) {
    if (code != hipSuccess) {
        std::fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

#define BLOCK_SIZE 128
#define TILE_SIZE_M 64  
#define TILE_SIZE_N 64  
#define HEAD_DIM 64     

__global__ void flash_attention_forward(
    const float* Q,    
    const float* K,
    const float* V,
    float* O,          
    const int seq_len,
    const float scale  
) {
    
    __shared__ float Q_tile[TILE_SIZE_M][HEAD_DIM];
    __shared__ float K_tile[TILE_SIZE_N][HEAD_DIM];
    __shared__ float V_tile[TILE_SIZE_N][HEAD_DIM];
    
    const int tid = threadIdx.x;
    const int batch = blockIdx.z;
    const int head = blockIdx.y;
    const int m_offset = blockIdx.x * TILE_SIZE_M;
    
    
    float O_local[HEAD_DIM] = {0.0f};
    float max_val = -INFINITY;
    float sum_exp = 0.0f;
    
    for (int n_start = 0; n_start < seq_len; n_start += TILE_SIZE_N) {
        
        if (tid < TILE_SIZE_M && (m_offset + tid) < seq_len) {
            #pragma unroll
            for (int d = 0; d < HEAD_DIM; ++d) {
                
                Q_tile[tid][d] = Q[((batch * gridDim.y + head) * seq_len + (m_offset + tid)) * HEAD_DIM + d];
            }
        }
        
        
        if (tid < TILE_SIZE_N && (n_start + tid) < seq_len) {
            #pragma unroll
            for (int d = 0; d < HEAD_DIM; ++d) {
                K_tile[tid][d] = K[((batch * gridDim.y + head) * seq_len + (n_start + tid)) * HEAD_DIM + d];
            }
        }
        __syncthreads();
        
        
        
        
        for (int n = 0; n < TILE_SIZE_N; ++n) {
            
            if (n_start + n < seq_len && tid < TILE_SIZE_M && (m_offset + tid) < seq_len) {
                float s = 0.0f;
                #pragma unroll
                for (int d = 0; d < HEAD_DIM; ++d) {
                    s += Q_tile[tid][d] * K_tile[n][d];
                }
                s *= scale;
                
                float old_max = max_val;
                max_val = fmaxf(max_val, s);
                sum_exp = sum_exp * expf(old_max - max_val) + expf(s - max_val);
            }
        }
        __syncthreads();
        
        if (tid < TILE_SIZE_N && (n_start + tid) < seq_len) {
            #pragma unroll
            for (int d = 0; d < HEAD_DIM; ++d) {
                V_tile[tid][d] = V[((batch * gridDim.y + head) * seq_len + (n_start + tid)) * HEAD_DIM + d];
            }
        }
        __syncthreads();
        
        
        
        for (int n = 0; n < TILE_SIZE_N; ++n) {
            if (n_start + n < seq_len && tid < TILE_SIZE_M && (m_offset + tid) < seq_len) {
                float s = 0.0f;
                #pragma unroll
                for (int d = 0; d < HEAD_DIM; ++d) {
                    s += Q_tile[tid][d] * K_tile[n][d];
                }
                s *= scale;
                float p = expf(s - max_val) / sum_exp;
                #pragma unroll
                for (int d = 0; d < HEAD_DIM; ++d) {
                    O_local[d] += p * V_tile[n][d];
                }
            }
        }
        __syncthreads();
    }
    
    if (tid < TILE_SIZE_M && (m_offset + tid) < seq_len) {
        #pragma unroll
        for (int d = 0; d < HEAD_DIM; ++d) {
            O[((batch * gridDim.y + head) * seq_len + (m_offset + tid)) * HEAD_DIM + d] = O_local[d];
        }
    }
}

void compute_flash_attention(
    const float* Q, const float* K, const float* V, float* O,
    int batch_size, int num_heads, int seq_len, int head_dim
) {
    dim3 grid_dim(
        (seq_len + TILE_SIZE_M - 1) / TILE_SIZE_M,  
        num_heads,                                  
        batch_size                                  
    );
    dim3 block_dim(BLOCK_SIZE);
    
    float scale = 1.0f / sqrtf(static_cast<float>(head_dim));
    
    flash_attention_forward<<<grid_dim, block_dim>>>( Q, K, V, O, seq_len, scale );
    CUDA_CHECK(hipDeviceSynchronize());
}
