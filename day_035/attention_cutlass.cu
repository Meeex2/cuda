#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>
#include <chrono>
#include <random>
#include <cutlass/cutlass.h>
#include <cutlass/gemm/device/gemm.h>

__global__ void softmax_kernel(const float* input, float* output, int num_rows, int num_cols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < num_rows) {
        float max_val = input[row * num_cols];
        for (int col = 1; col < num_cols; col++) {
            max_val = fmaxf(max_val, input[row * num_cols + col]);
        }
        float sum = 0.0f;
        for (int col = 0; col < num_cols; col++) {
            output[row * num_cols + col] = expf(input[row * num_cols + col] - max_val);
            sum += output[row * num_cols + col];
        }
        for (int col = 0; col < num_cols; col++) {
            output[row * num_cols + col] /= sum;
        }
    }
}

