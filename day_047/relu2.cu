#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <chrono>
#include <cstring>  // Include for memcpy

// CUDA Kernel for ReLU²
__global__ void relu2_kernel(float* data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float x = data[idx];
        data[idx] = (x > 0) ? x * x : 0.0f;  // ReLU²: x² if x > 0, else 0
    }
}

// CPU Implementation of ReLU² for Comparison
void relu2_cpu(float* data, int size) {
    for (int i = 0; i < size; ++i) {
        float x = data[i];
        data[i] = (x > 0) ? x * x : 0.0f;  // ReLU²: x² if x > 0, else 0
    }
}

